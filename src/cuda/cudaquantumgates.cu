#include "hip/hip_runtime.h"
/*
Copyright (c) Microsoft. All Rights Reserved.
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h> 
#include <cstdlib>
#include <cstdint>
#include <vector>

#include <math.h>
#include <time.h>
#include <numeric>
#include <sstream>
#include <iostream>

#include <chrono>

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR
#include <memory>


__global__ void isqubitclassical(hipDoubleComplex* __restrict__ vector, int64_t number_of_elements,double eps, bool *haves_out, int64_t offset, int64_t mask) {
    const int64_t thIdx = threadIdx.x;
    const int64_t blockSize = blockDim.x;
    const int64_t gthIdx = thIdx + blockIdx.x*blockSize;
    
    bool have0 = false;
    bool have1 = false;
    const int64_t gridSize = blockSize*gridDim.x;

    for (int64_t elemId = gthIdx; elemId < number_of_elements / 2; elemId += gridSize)
    {
        {
            int64_t j = elemId % offset;
            int64_t i = ((elemId & mask) << 1);
            int64_t pos0 = i + j;
            int64_t pos1 = i + j + offset;
            double value0 = sqrt(vector[pos0].x * vector[pos0].x + vector[pos0].y * vector[pos0].y);
            double value1 = sqrt(vector[pos1].x * vector[pos1].x + vector[pos1].y * vector[pos1].y);

            have0 |= (value0 >= eps);
            have1 |= (value1 >= eps);
        }
    }
    
    __shared__ bool have0s[1024];
    __shared__ bool have1s[1024];
    have0s[thIdx] = have0;
    have1s[thIdx] = have1;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) 
    { //uniform
        if (thIdx<size)
        {
            have0s[thIdx] += have0s[thIdx+size];
            have1s[thIdx] += have1s[thIdx+size];
        }
        __syncthreads();
    }
    if (thIdx == 0)
    {
        haves_out[blockIdx.x * 2] = have0s[0];
        haves_out[blockIdx.x * 2 + 1] = have1s[0];
    }
}

__global__ void my_custom_init(hipDoubleComplex* __restrict__ vector, int64_t number_of_elements, int64_t bit_value) {
	int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
	if(i < number_of_elements) 
    {
        if (i == bit_value)
        {
            vector[i] = make_hipDoubleComplex( 1., 0.);
        }
        else
        {
            vector[i] = make_hipDoubleComplex( 0., 0.);
        }
	}
}

__global__ void get_qubit_value(hipDoubleComplex* __restrict__ vector, const int64_t number_of_elements, int64_t mask, double eps, int* result)
{
    int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    if(i < number_of_elements) 
    {
        double value = sqrt(vector[i].x * vector[i].x + vector[i].y * vector[i].y);
        if (value > eps) 
        {
            // only one thread shall hit this
            result[0] = (i & mask ? 1 : 0);
        }
    }
}

__global__ void collapse_one_bit(hipDoubleComplex* __restrict__ vector, int64_t number_of_elements, uint64_t state, uint64_t mask) {
	int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
	if(i < number_of_elements) 
    {
        if ((((uint64_t)i) & mask) != state) vector[i] = make_hipDoubleComplex( 0., 0.);
	}
}

static custatevecPauli_t  E_Paulis[4] = {CUSTATEVEC_PAULI_I, CUSTATEVEC_PAULI_X, CUSTATEVEC_PAULI_Z, CUSTATEVEC_PAULI_Y};


class CTiming
{
  public:
    CTiming(int64_t& totalIn) : total_(totalIn)
    {
        start_ = std::chrono::duration_cast<std::chrono::microseconds>(
                    std::chrono::high_resolution_clock::now().time_since_epoch())
                    .count();
    }

    ~CTiming()
    {
        auto end = std::chrono::duration_cast<std::chrono::microseconds>(
                       std::chrono::high_resolution_clock::now().time_since_epoch())
                       .count();
        total_ += (end - start_) / 1000;
    }
private:
    int64_t start_;
    int64_t& total_;
};

class AutoHandle
{
    public:
        AutoHandle(custatevecHandle_t& handle): handle_(handle)
        {
            if (handle_ == 0)
            {
                HANDLE_ERROR( custatevecCreate(&handle_) );
            }
        }
        ~AutoHandle()
        {
            if (handle_ != 0)
            {
                HANDLE_ERROR( custatevecDestroy(handle_) );
                handle_ = 0;
            }
        }
    private:
        custatevecHandle_t& handle_;
};

class CudaContext
{
    public:
        CudaContext(int nQubits): nQubits_(0), handle_(0), wfn_(nullptr)
        , measure_ms_(0), apply_ms_(0), create_ms_(0), exp_ms_(0), release_ms_(0), collapse_ms_(0), classical_ms_(0), get_ms_(0), applys_(0), measures_(0)
        {
            if (nQubits >0)
            {
                create_qubits(nQubits);
            }

            //HANDLE_ERROR( custatevecCreate(&handle_) );
        }

        ~CudaContext()
        {
            if (handle_ != 0)
            {
                HANDLE_ERROR( custatevecDestroy(handle_) );
            }
            
            if (wfn_ != nullptr)
            {
                HANDLE_CUDA_ERROR( hipFree(wfn_) );
            }

            std::cout<<"measure:"<<measure_ms_<<", apply:"<<apply_ms_<<", create:"<<create_ms_<<", exp:"<<exp_ms_<<", release:"<<release_ms_<<", collapse:"<<collapse_ms_<<", classical:"<<classical_ms_<<", get:"<<get_ms_<<std::endl;
            std::cout<<"measure count: "<<measures_<<", apply count:"<<applys_<<std::endl;
        }

        inline void release_qubits()
        {
            if (wfn_ != nullptr)
            {
                CTiming timer(release_ms_);
                HANDLE_CUDA_ERROR( hipFree(wfn_) );
                wfn_ = nullptr;
                nQubits_ = 0;
            }
        }
        inline int num_qubits() const
        {
            return nQubits_;
        }

        inline custatevecHandle_t handle() const
        {
            return handle_;
        }

        inline hipDoubleComplex* wfn() const
        {
            return wfn_;
        }

        inline void create_qubits(int num_qubits)
        {
            if (num_qubits != nQubits_)
            {
                CTiming timer(create_ms_);
                if (nQubits_ != 0)
                {
                    HANDLE_CUDA_ERROR( hipFree(wfn_) );
                    wfn_ = nullptr;
                }
                nQubits_ = num_qubits;

                if (num_qubits > 0)
                {
                    const int64_t nSvSize    = (1LL << num_qubits);
                    HANDLE_CUDA_ERROR(hipMalloc((void**)&wfn_, nSvSize * sizeof(hipDoubleComplex)));
                    const int64_t blockSize = 1024;
                    const int64_t gridSize = (nSvSize + blockSize - 1) / blockSize;

                    my_custom_init<<<gridSize, blockSize>>>(wfn_, nSvSize, 0);
                }
            }
        }

        inline int get_value(int64_t mask, double eps)
        {
            const int64_t nSvSize    = (1LL << nQubits_);
            const int64_t blockSize = 1024;
            const int64_t gridSize = (nSvSize + blockSize - 1) / blockSize;
            int * device_out;
            HANDLE_CUDA_ERROR(hipMalloc((void**)&device_out, sizeof(int)));
            get_qubit_value<<<gridSize, blockSize>>>(wfn_, nSvSize, (int64_t)mask, eps, device_out);
            int result;
            HANDLE_CUDA_ERROR(hipMemcpy(&result, device_out, sizeof(int), hipMemcpyDeviceToHost));
            HANDLE_CUDA_ERROR(hipFree(device_out));
            return (result == 1 || result == 0) ? result : 2;
        }

        inline void collapse(uint64_t mask, uint64_t state)
        {
            CTiming timer(collapse_ms_);
            const int64_t nSvSize    = (1LL << nQubits_);
            const int64_t blockSize = 1024;
            const int64_t gridSize = (nSvSize + blockSize - 1) / blockSize;

            collapse_one_bit<<<gridSize, blockSize>>>(wfn_, nSvSize, state, mask);
        }

        inline void isclassical(double eps, bool *haves_out, int64_t offset, int64_t mask, int64_t num_blocks)
        {
            CTiming timer(classical_ms_);
            const int64_t nSvSize    = (1LL << nQubits_);
            const int64_t blockSize = 1024;
            bool *out_buf_device = nullptr;
            const size_t output_size = num_blocks * sizeof(bool) * 2;
            HANDLE_CUDA_ERROR(hipMalloc((void**)&out_buf_device, output_size));
            isqubitclassical<<<num_blocks, blockSize>>>(wfn_, nSvSize, eps, out_buf_device, offset, mask);
            HANDLE_CUDA_ERROR(hipMemcpy(haves_out, out_buf_device, output_size, hipMemcpyDeviceToHost));
            HANDLE_CUDA_ERROR(hipFree(out_buf_device));
        }

        inline void increase_one_qubit()
        {
            create_qubits(nQubits_ + 1);
        }

        inline void apply_controlled_exp(double theta, const int* paulis, const int32_t *targets, 
        const uint32_t nTargets, const int32_t *controls, const int32_t *controlBitValues, const uint32_t nControls)
        {
            CTiming timer(exp_ms_);
            std::vector<custatevecPauli_t> e_paulis(nTargets);
            for (uint32_t i = 0; i < nTargets; i++)
            {
                e_paulis[i] = E_Paulis[paulis[i]];
            }
            AutoHandle autohandle(handle_);
            HANDLE_ERROR(custatevecApplyExp(handle_, wfn_, HIP_C_64F, (uint32_t)nQubits_, 
            theta, e_paulis.data(), targets, nTargets, controls, controlBitValues, nControls));
        }

        inline void meansure_zbasis(int32_t *parity, const int32_t *basis_bits, uint32_t nbasis_bits, double randnum, bool collapse)
        {
            CTiming timer(measure_ms_);
            measures_++;
            AutoHandle autohandle(handle_);
            custatevecHandle_t handle = handle_;
            HANDLE_ERROR(custatevecMeasureOnZBasis(handle, wfn(), HIP_C_64F, num_qubits(), parity, basis_bits, nbasis_bits, randnum, 
            collapse ? CUSTATEVEC_COLLAPSE_NORMALIZE_AND_ZERO : CUSTATEVEC_COLLAPSE_NONE));
        }

        inline void apply_gate(const hipDoubleComplex* matrix, int nControls, int nTargets, const int* targets, const int* controls)
        {
            CTiming timer(apply_ms_);
            applys_++;
            size_t extraWorkspaceSizeInBytes = 0;
            void* extraWorkspace = nullptr;
            {
            AutoHandle autohandle(handle_);
            custatevecHandle_t handle = handle_;
            int adjoint = 0;

            // check the size of external workspace
            HANDLE_ERROR( custatevecApplyMatrix_bufferSize(
                        handle, HIP_C_64F, num_qubits(), matrix, HIP_C_64F, CUSTATEVEC_MATRIX_LAYOUT_ROW,
                        adjoint, nTargets, nControls, CUSTATEVEC_COMPUTE_64F, &extraWorkspaceSizeInBytes) );

            // allocate external workspace if necessary
            if (extraWorkspaceSizeInBytes > 0)
                HANDLE_CUDA_ERROR( hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes) );

            //auto start_ms = std::chrono::duration_cast<std::chrono::microseconds>(
            //         std::chrono::high_resolution_clock::now().time_since_epoch())
            //         .count();

            // apply gate
            HANDLE_ERROR( custatevecApplyMatrix(
                        handle, wfn(), HIP_C_64F, num_qubits(), matrix, HIP_C_64F,
                        CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, targets, nTargets, controls, nControls, 
                        nullptr, CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes) );

            //auto end_ms = std::chrono::duration_cast<std::chrono::microseconds>(
            //         std::chrono::high_resolution_clock::now().time_since_epoch())
			//			.count();
		    // std::cout<<"custatevecApplyMatrix time:"<<(end_ms - start_ms)/1000.<<std::endl;
            }
            if (extraWorkspaceSizeInBytes)
                HANDLE_CUDA_ERROR( hipFree(extraWorkspace) );
        }

    private:
        int nQubits_;
        custatevecHandle_t handle_;
        hipDoubleComplex *wfn_;
        int64_t measure_ms_;
        int64_t apply_ms_;
        int64_t create_ms_;
        int64_t exp_ms_;
        int64_t release_ms_;
        int64_t collapse_ms_;
        int64_t classical_ms_;
        int64_t get_ms_;
        int64_t applys_;
        int64_t measures_;
};


extern "C"
{
    void* create_cuquantum_context()
    {
        void* pInstance = new CudaContext(0);
        return pInstance;
    }
    void  free_cuquantum_context(void* handle)
    {
        if (handle != nullptr)
        {
            CudaContext* pContext = reinterpret_cast<CudaContext*>(handle);
            delete pContext;
        }
    }
    void apply_cuquantum_gate(void* handle, const double* matrix, int nControls, int nTargets, const int* targets, const int* controls)
    {
        CudaContext* pContext = reinterpret_cast<CudaContext*>(handle);
        const hipDoubleComplex* pMatrix = reinterpret_cast<const hipDoubleComplex*>(matrix);
        pContext->apply_gate(pMatrix, nControls, nTargets, targets, controls);
    }
    void create_cuquantum_bits(void* handle, int num_qubits)
    {
        CudaContext* pContext = reinterpret_cast<CudaContext*>(handle);
        pContext->create_qubits(num_qubits);
    }

    void increase_cuquantum_bit(void* handle)
    {
        CudaContext* pContext = reinterpret_cast<CudaContext*>(handle);
        pContext->increase_one_qubit();
    }

    void meansure_cuquantum_zbasis(void* handle, int32_t *parity, const int32_t *basis_bits, uint32_t nbasis_bits, double randnum, bool collapse)
    {
        CudaContext* pContext = reinterpret_cast<CudaContext*>(handle);
        pContext->meansure_zbasis(parity, basis_bits, nbasis_bits, randnum, collapse);
    }

    void collapse_cuquantum(void* handle, uint64_t mask, uint64_t state)
    {
        CudaContext* p_context = reinterpret_cast<CudaContext*>(handle);
        p_context->collapse(mask, state);
    }

    void apply_cuquantum_controlled_exp(void* handle, double theta, const int* paulis, const int32_t *targets, 
        const uint32_t nTargets, const int32_t *controls, const int32_t *controlBitValues, const uint32_t nControls)
    {
        CudaContext* p_context = reinterpret_cast<CudaContext*>(handle);
        p_context->apply_controlled_exp(theta, paulis, targets, nTargets, controls, controlBitValues, nControls);
    }

    void release_cuquantum_bits(void* handle)
    {
        CudaContext* p_context = reinterpret_cast<CudaContext*>(handle);
        p_context->release_qubits();
    }

    void is_cuquantum_classical(void* handle, double eps, bool *haves_out, int64_t offset, int64_t mask, int64_t num_blocks)
    {
        CudaContext* p_context = reinterpret_cast<CudaContext*>(handle);
        p_context->isclassical(eps, haves_out, offset, mask, num_blocks);
    }

    int get_cuquantum_value(void* handle, int64_t mask, double eps)
    {
        CudaContext* p_context = reinterpret_cast<CudaContext*>(handle);
        return p_context->get_value(mask, eps);
    }
}

