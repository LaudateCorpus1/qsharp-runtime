#include "hip/hip_runtime.h"
/*
Copyright (c) Microsoft. All Rights Reserved.
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h> 
#include <cstdlib>
#include <cstdint>
#include <vector>

#include <math.h>
#include <time.h>
#include <numeric>
#include <sstream>
#include <iostream>

#include <chrono>

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR
#include <memory>

__global__ void my_custom_init(hipDoubleComplex* __restrict__ vector, int64_t number_of_elements, int64_t bit_value) {
	int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
	if(i < number_of_elements) 
    {
        if (i == bit_value)
        {
            vector[i] = make_hipDoubleComplex( 1., 0.);
        }
        else
        {
            vector[i] = make_hipDoubleComplex( 0., 0.);
        }
	}
}

__global__ void collapse_one_bit(hipDoubleComplex* __restrict__ vector, int64_t number_of_elements, uint64_t state, uint64_t mask) {
	int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
	if(i < number_of_elements) 
    {
        if ((((uint64_t)i) & mask) != state) vector[i] = make_hipDoubleComplex( 0., 0.);
	}
}

static custatevecPauli_t  E_Paulis[4] = {CUSTATEVEC_PAULI_I, CUSTATEVEC_PAULI_X, CUSTATEVEC_PAULI_Z, CUSTATEVEC_PAULI_Y};


class CTiming
{
  public:
    CTiming(int64_t& totalIn) : total_(totalIn)
    {
        start_ = std::chrono::duration_cast<std::chrono::microseconds>(
                    std::chrono::high_resolution_clock::now().time_since_epoch())
                    .count();
    }

    ~CTiming()
    {
        auto end = std::chrono::duration_cast<std::chrono::microseconds>(
                       std::chrono::high_resolution_clock::now().time_since_epoch())
                       .count();
        total_ += (end - start_) / 1000;
    }
private:
    int64_t start_;
    int64_t& total_;
};

class CudaContext
{
    public:
        CudaContext(int nQubits): nQubits_(0), handle_(0), wfn_(nullptr)
        , measure_ms_(0), apply_ms_(0), create_ms_(0), exp_ms_(0), release_ms_(0), collapse_ms_(0), applys_(0), measures_(0)
        {
            if (nQubits >0)
            {
                create_qubits(nQubits);
            }

            HANDLE_ERROR( custatevecCreate(&handle_) );
        }

        ~CudaContext()
        {
            if (handle_ != 0)
            {
                HANDLE_ERROR( custatevecDestroy(handle_) );
            }
            
            if (wfn_ != nullptr)
            {
                HANDLE_CUDA_ERROR( hipFree(wfn_) );
            }

            std::cout<<"measure:"<<measure_ms_<<", apply:"<<apply_ms_<<", create:"<<create_ms_<<", exp:"<<exp_ms_<<", release:"<<release_ms_<<", collapse:"<<collapse_ms_<<std::endl;
            std::cout<<"measure count: "<<measures_<<", apply count:"<<applys_<<std::endl;
        }

        inline void release_qubits()
        {
            if (wfn_ != nullptr)
            {
                CTiming timer(release_ms_);
                HANDLE_CUDA_ERROR( hipFree(wfn_) );
                wfn_ = nullptr;
                nQubits_ = 0;
            }
        }
        inline int num_qubits() const
        {
            return nQubits_;
        }

        inline custatevecHandle_t handle() const
        {
            return handle_;
        }

        inline hipDoubleComplex* wfn() const
        {
            return wfn_;
        }

        inline void create_qubits(int num_qubits)
        {
            if (num_qubits != nQubits_)
            {
                CTiming timer(create_ms_);
                if (nQubits_ != 0)
                {
                    HANDLE_CUDA_ERROR( hipFree(wfn_) );
                    wfn_ = nullptr;
                }
                nQubits_ = num_qubits;

                if (num_qubits > 0)
                {
                    const int64_t nSvSize    = (1LL << num_qubits);
                    HANDLE_CUDA_ERROR(hipMalloc((void**)&wfn_, nSvSize * sizeof(hipDoubleComplex)));
                    //const int64_t blockSize = 1024;
                    //const int64_t gridSize = (nSvSize + blockSize - 1) / blockSize;

                    //my_custom_init<<<gridSize, blockSize>>>(wfn_, nSvSize, 0);
                }
            }
        }

        inline void collapse(uint64_t mask, uint64_t state)
        {
            CTiming timer(collapse_ms_);
            const int64_t nSvSize    = (1LL << nQubits_);
            const int64_t blockSize = 1024;
            const int64_t gridSize = (nSvSize + blockSize - 1) / blockSize;

            collapse_one_bit<<<gridSize, blockSize>>>(wfn_, nSvSize, state, mask);
        }

        inline void increase_one_qubit()
        {
            create_qubits(nQubits_ + 1);
        }

        inline void apply_controlled_exp(double theta, const int* paulis, const int32_t *targets, 
        const uint32_t nTargets, const int32_t *controls, const int32_t *controlBitValues, const uint32_t nControls)
        {
            CTiming timer(exp_ms_);
            std::vector<custatevecPauli_t> e_paulis(nTargets);
            for (uint32_t i = 0; i < nTargets; i++)
            {
                e_paulis[i] = E_Paulis[paulis[i]];
            }
            HANDLE_ERROR(custatevecApplyExp(handle_, wfn_, HIP_C_64F, (uint32_t)nQubits_, 
            theta, e_paulis.data(), targets, nTargets, controls, controlBitValues, nControls));
        }

        inline void meansure_zbasis(int32_t *parity, const int32_t *basis_bits, uint32_t nbasis_bits, double randnum, bool collapse)
        {
            CTiming timer(measure_ms_);
            measures_++;
            custatevecHandle_t handle = handle_;
            HANDLE_ERROR(custatevecMeasureOnZBasis(handle, wfn(), HIP_C_64F, num_qubits(), parity, basis_bits, nbasis_bits, randnum, 
            collapse ? CUSTATEVEC_COLLAPSE_NORMALIZE_AND_ZERO : CUSTATEVEC_COLLAPSE_NONE));
        }

        inline void apply_gate(const hipDoubleComplex* matrix, int nControls, int nTargets, const int* targets, const int* controls)
        {
            CTiming timer(apply_ms_);
            applys_++;
            custatevecHandle_t handle = handle_;
            int adjoint = 0;

            void* extraWorkspace = nullptr;
            size_t extraWorkspaceSizeInBytes = 0;

            // check the size of external workspace
            HANDLE_ERROR( custatevecApplyMatrix_bufferSize(
                        handle, HIP_C_64F, num_qubits(), matrix, HIP_C_64F, CUSTATEVEC_MATRIX_LAYOUT_ROW,
                        adjoint, nTargets, nControls, CUSTATEVEC_COMPUTE_64F, &extraWorkspaceSizeInBytes) );

            // allocate external workspace if necessary
            if (extraWorkspaceSizeInBytes > 0)
                HANDLE_CUDA_ERROR( hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes) );

            auto start_ms = std::chrono::duration_cast<std::chrono::microseconds>(
                     std::chrono::high_resolution_clock::now().time_since_epoch())
                     .count();

            // apply gate
            HANDLE_ERROR( custatevecApplyMatrix(
                        handle, wfn(), HIP_C_64F, num_qubits(), matrix, HIP_C_64F,
                        CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, targets, nTargets, controls, nControls, 
                        nullptr, CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes) );

            auto end_ms = std::chrono::duration_cast<std::chrono::microseconds>(
                     std::chrono::high_resolution_clock::now().time_since_epoch())
						.count();
		    std::cout<<"loop "<<i<<", time:"<<(end_ms - start_ms)/1000.<<std::endl;

            if (extraWorkspaceSizeInBytes)
                HANDLE_CUDA_ERROR( hipFree(extraWorkspace) );
        }

    private:
        int nQubits_;
        custatevecHandle_t handle_;
        hipDoubleComplex *wfn_;
        int64_t measure_ms_;
        int64_t apply_ms_;
        int64_t create_ms_;
        int64_t exp_ms_;
        int64_t release_ms_;
        int64_t collapse_ms_;
        int64_t applys_;
        int64_t measures_;
};


extern "C"
{
    void* create_cuquantum_context()
    {
        void* pInstance = new CudaContext(0);
        return pInstance;
    }
    void  free_cuquantum_context(void* handle)
    {
        if (handle != nullptr)
        {
            CudaContext* pContext = reinterpret_cast<CudaContext*>(handle);
            delete pContext;
        }
    }
    void apply_cuquantum_gate(void* handle, const double* matrix, int nControls, int nTargets, const int* targets, const int* controls)
    {
        CudaContext* pContext = reinterpret_cast<CudaContext*>(handle);
        const hipDoubleComplex* pMatrix = reinterpret_cast<const hipDoubleComplex*>(matrix);
        pContext->apply_gate(pMatrix, nControls, nTargets, targets, controls);
    }
    void create_cuquantum_bits(void* handle, int num_qubits)
    {
        CudaContext* pContext = reinterpret_cast<CudaContext*>(handle);
        pContext->create_qubits(num_qubits);
    }

    void increase_cuquantum_bit(void* handle)
    {
        CudaContext* pContext = reinterpret_cast<CudaContext*>(handle);
        pContext->increase_one_qubit();
    }

    void meansure_cuquantum_zbasis(void* handle, int32_t *parity, const int32_t *basis_bits, uint32_t nbasis_bits, double randnum, bool collapse)
    {
        CudaContext* pContext = reinterpret_cast<CudaContext*>(handle);
        pContext->meansure_zbasis(parity, basis_bits, nbasis_bits, randnum, collapse);
    }

    void collapse_cuquantum(void* handle, uint64_t mask, uint64_t state)
    {
        CudaContext* p_context = reinterpret_cast<CudaContext*>(handle);
        p_context->collapse(mask, state);
    }

    void apply_cuquantum_controlled_exp(void* handle, double theta, const int* paulis, const int32_t *targets, 
        const uint32_t nTargets, const int32_t *controls, const int32_t *controlBitValues, const uint32_t nControls)
    {
        CudaContext* p_context = reinterpret_cast<CudaContext*>(handle);
        p_context->apply_controlled_exp(theta, paulis, targets, nTargets, controls, controlBitValues, nControls);
    }

    void release_cuquantum_bits(void* handle)
    {
        CudaContext* p_context = reinterpret_cast<CudaContext*>(handle);
        p_context->release_qubits();
    }
}

