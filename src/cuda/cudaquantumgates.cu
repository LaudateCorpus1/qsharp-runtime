#include "hip/hip_runtime.h"
/*
Copyright (c) Microsoft. All Rights Reserved.
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h> 
#include <cstdlib>
#include <cstdint>
#include <vector>

#include <math.h>
#include <time.h>
#include <numeric>
#include <sstream>
#include <iostream>

#include <chrono>

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR
#include <memory>

__global__ void my_custom_init(hipDoubleComplex* __restrict__ vector, int64_t number_of_elements, int64_t bit_value) {
	int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
	if(i < number_of_elements) 
    {
        if (i == bit_value)
        {
            vector[i] = make_hipDoubleComplex( 1., 0.);
        }
        else
        {
            vector[i] = make_hipDoubleComplex( 0., 0.);
        }
	}
}

__global__ void collapse_one_bit(hipDoubleComplex* __restrict__ vector, int64_t number_of_elements, uint64_t state, uint64_t mask) {
	int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
	if(i < number_of_elements) 
    {
        if ((((uint64_t)i) & mask) != state) vector[i] = make_hipDoubleComplex( 0., 0.);
	}
}

static custatevecPauli_t  E_Paulis[4] = {CUSTATEVEC_PAULI_I, CUSTATEVEC_PAULI_X, CUSTATEVEC_PAULI_Z, CUSTATEVEC_PAULI_Y};

class CudaContext
{
    public:
        CudaContext(int nQubits): nQubits_(0), handle_(0), wfn_(nullptr)
        {
            if (nQubits >0)
            {
                create_qubits(nQubits);
            }

            HANDLE_ERROR( custatevecCreate(&handle_) );
        }

        ~CudaContext()
        {
            if (handle_ != 0)
            {
                HANDLE_ERROR( custatevecDestroy(handle_) );
            }
            
            if (wfn_ != nullptr)
            {
                HANDLE_CUDA_ERROR( hipFree(wfn_) );
            }
        }
        inline int num_qubits() const
        {
            return nQubits_;
        }

        inline custatevecHandle_t handle() const
        {
            return handle_;
        }

        inline hipDoubleComplex* wfn() const
        {
            return wfn_;
        }

        inline void create_qubits(int num_qubits)
        {
            if (num_qubits != nQubits_)
            {
                if (nQubits_ != 0)
                {
                    HANDLE_CUDA_ERROR( hipFree(wfn_) );
                    wfn_ = nullptr;
                }
                nQubits_ = num_qubits;

                if (num_qubits > 0)
                {
                    const int64_t nSvSize    = (1LL << num_qubits);
                    HANDLE_CUDA_ERROR(hipMalloc((void**)&wfn_, nSvSize * sizeof(hipDoubleComplex)));
                    const int64_t blockSize = 1024;
                    const int64_t gridSize = (nSvSize + blockSize - 1) / blockSize;

                    my_custom_init<<<gridSize, blockSize>>>(wfn_, nSvSize, 0);
                }
            }
        }

        inline void collapse(uint64_t mask, uint64_t state)
        {
            const int64_t nSvSize    = (1LL << nQubits_);
            const int64_t blockSize = 1024;
            const int64_t gridSize = (nSvSize + blockSize - 1) / blockSize;

            collapse_one_bit<<<gridSize, blockSize>>>(wfn_, nSvSize, state, mask);
        }

        inline void increase_one_qubit()
        {
            create_qubits(nQubits_ + 1);
        }

        inline void apply_controlled_exp(double theta, const int* paulis, const int32_t *targets, 
        const uint32_t nTargets, const int32_t *controls, const int32_t *controlBitValues, const uint32_t nControls)
        {
            std::vector<custatevecPauli_t> e_paulis(nTargets);
            for (uint32_t i = 0; i < nTargets; i++)
            {
                e_paulis[i] = E_Paulis[paulis[i]];
            }
            HANDLE_ERROR(custatevecApplyExp(handle_, wfn_, HIP_C_64F, (uint32_t)nQubits_, 
            theta, e_paulis.data(), targets, nTargets, controls, controlBitValues, nControls));
        }
    private:
        int nQubits_;
        custatevecHandle_t handle_;
        hipDoubleComplex *wfn_;
};

void apply_gate(const CudaContext* pContext, const hipDoubleComplex* matrix, int nControls, int nTargets, const int* targets, const int* controls)
{
    custatevecHandle_t handle = pContext->handle();
    int adjoint = 0;

    void* extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    // check the size of external workspace
    HANDLE_ERROR( custatevecApplyMatrix_bufferSize(
                  handle, HIP_C_64F, pContext->num_qubits(), matrix, HIP_C_64F, CUSTATEVEC_MATRIX_LAYOUT_ROW,
                  adjoint, nTargets, nControls, CUSTATEVEC_COMPUTE_64F, &extraWorkspaceSizeInBytes) );

    // allocate external workspace if necessary
    if (extraWorkspaceSizeInBytes > 0)
        HANDLE_CUDA_ERROR( hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes) );


    // apply gate
    HANDLE_ERROR( custatevecApplyMatrix(
                handle, pContext->wfn(), HIP_C_64F, pContext->num_qubits(), matrix, HIP_C_64F,
                CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, targets, nTargets, controls, nControls, 
                nullptr, CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes) );

    if (extraWorkspaceSizeInBytes)
        HANDLE_CUDA_ERROR( hipFree(extraWorkspace) );
}

void meansure_zbasis(const CudaContext* p_context, int32_t *parity, const int32_t *basis_bits, uint32_t nbasis_bits, double randnum, bool collapse)
{
    custatevecHandle_t handle = p_context->handle();
    HANDLE_ERROR(custatevecMeasureOnZBasis(handle, p_context->wfn(), HIP_C_64F, p_context->num_qubits(), parity, basis_bits, nbasis_bits, randnum, 
    collapse ? CUSTATEVEC_COLLAPSE_NORMALIZE_AND_ZERO : CUSTATEVEC_COLLAPSE_NONE));
}

extern "C"
{
    void* create_cuquantum_context()
    {
        void* pInstance = new CudaContext(0);
        return pInstance;
    }
    void  free_cuquantum_context(void* handle)
    {
        if (handle != nullptr)
        {
            CudaContext* pContext = reinterpret_cast<CudaContext*>(handle);
            delete pContext;
        }
    }
    void apply_cuquantum_gate(const void* handle, const double* matrix, int nControls, int nTargets, const int* targets, const int* controls)
    {
        const CudaContext* pContext = reinterpret_cast<const CudaContext*>(handle);
        const hipDoubleComplex* pMatrix = reinterpret_cast<const hipDoubleComplex*>(matrix);
        apply_gate(pContext, pMatrix, nControls, nTargets, targets, controls);
    }
    void create_cuquantum_bits(void* handle, int num_qubits)
    {
        CudaContext* pContext = reinterpret_cast<CudaContext*>(handle);
        pContext->create_qubits(num_qubits);
    }

    void increase_cuquantum_bit(void* handle)
    {
        CudaContext* pContext = reinterpret_cast<CudaContext*>(handle);
        pContext->increase_one_qubit();
    }

    void meansure_cuquantum_zbasis(void* handle, int32_t *parity, const int32_t *basis_bits, uint32_t nbasis_bits, double randnum, bool collapse)
    {
        CudaContext* pContext = reinterpret_cast<CudaContext*>(handle);
        meansure_zbasis(pContext, parity, basis_bits, nbasis_bits, randnum, collapse);
    }

    void collapse_cuquantum(void* handle, uint64_t mask, uint64_t state)
    {
        CudaContext* p_context = reinterpret_cast<CudaContext*>(handle);
        p_context->collapse(mask, state);
    }

    void apply_cuquantum_controlled_exp(void* handle, double theta, const int* paulis, const int32_t *targets, 
        const uint32_t nTargets, const int32_t *controls, const int32_t *controlBitValues, const uint32_t nControls)
    {
        CudaContext* p_context = reinterpret_cast<CudaContext*>(handle);
        p_context->apply_controlled_exp(theta, paulis, targets, nTargets, controls, controlBitValues, nControls);
    }
}

