#include "hip/hip_runtime.h"
/*
Copyright (c) Microsoft. All Rights Reserved.
 */

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <custatevec.h>       // custatevecApplyMatrix
#include <stdio.h>            // printf
#include <stdlib.h> 
#include <cstdlib>

#include <math.h>
#include <time.h>
#include <numeric>
#include <sstream>
#include <iostream>

#include <chrono>

#include "helper.hpp"         // HANDLE_ERROR, HANDLE_CUDA_ERROR
#include <memory>

__global__ void my_custom_init(hipDoubleComplex* __restrict__ vector, int64_t number_of_elements, int64_t bit_value) {
	int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
	if(i < number_of_elements) 
    {
        if (i == bit_value)
        {
            vector[i] = make_hipDoubleComplex( 1., 0.);
        }
        else
        {
            vector[i] = make_hipDoubleComplex( 0., 0.);
        }
	}
}

__global__ void my_maxtrix_init(hipDoubleComplex* __restrict__ vector, int64_t number_of_elements) {
	int64_t i = static_cast<int64_t>(blockIdx.x) * blockDim.x + threadIdx.x;
	if(i < number_of_elements) 
    {
        vector[i] = make_hipDoubleComplex( (double)i, 0.);
	}
}

void run_gate(const hipDoubleComplex* matrix, int nControls, int nTargets, int adjoint, const int* targets, const int* controls, int nIndexBits)
{
    const int64_t nSvSize    = (1LL << nIndexBits);
    const int loops = 3;

    hipDoubleComplex *d_sv;
    HANDLE_CUDA_ERROR(hipMalloc((void**)&d_sv, nSvSize * sizeof(hipDoubleComplex)));

    my_custom_init<<<84, 128>>>(d_sv, nSvSize, 0) ;

    hipDoubleComplex *d_matrix = nullptr;
    if (matrix == nullptr)
    {
        const int64_t nMSize    = (1LL << nTargets);
        HANDLE_CUDA_ERROR(hipMalloc((void**)&d_matrix, nMSize * nMSize * sizeof(hipDoubleComplex)));
        my_maxtrix_init<<<84, 128>>>(d_matrix, nMSize) ;
    }

    //----------------------------------------------------------------------------------------------

    // custatevec handle initialization
    custatevecHandle_t handle;
    HANDLE_ERROR( custatevecCreate(&handle) );

    void* extraWorkspace = nullptr;
    size_t extraWorkspaceSizeInBytes = 0;

    // check the size of external workspace
    HANDLE_ERROR( custatevecApplyMatrix_bufferSize(
                  handle, HIP_C_64F, nIndexBits, matrix == nullptr ? d_matrix : matrix, HIP_C_64F, CUSTATEVEC_MATRIX_LAYOUT_ROW,
                  adjoint, nTargets, nControls, CUSTATEVEC_COMPUTE_64F, &extraWorkspaceSizeInBytes) );

    // allocate external workspace if necessary
    if (extraWorkspaceSizeInBytes > 0)
        HANDLE_CUDA_ERROR( hipMalloc(&extraWorkspace, extraWorkspaceSizeInBytes) );

    for(int i = 0; i < loops; i++)
    {
        // apply gate
        HANDLE_ERROR( custatevecApplyMatrix(
                    handle, d_sv, HIP_C_64F, nIndexBits, matrix == nullptr ? d_matrix : matrix, HIP_C_64F,
                    CUSTATEVEC_MATRIX_LAYOUT_ROW, adjoint, targets, nTargets, controls, nControls, 
                    nullptr, CUSTATEVEC_COMPUTE_64F, extraWorkspace, extraWorkspaceSizeInBytes) );
    }
    // destroy handle
    HANDLE_ERROR( custatevecDestroy(handle) );

    //----------------------------------------------------------------------------------------------


    HANDLE_CUDA_ERROR( hipFree(d_sv) );
    if (extraWorkspaceSizeInBytes)
        HANDLE_CUDA_ERROR( hipFree(extraWorkspace) );

    if (d_matrix != nullptr)
    {
        HANDLE_CUDA_ERROR( hipFree(d_matrix) );
    }
}

class Timing
{
    public:
        Timing(std::string name, uint32_t times = 1): name_(name), times_(times)
        {
            start_ = std::chrono::duration_cast<std::chrono::microseconds>(
                     std::chrono::high_resolution_clock::now().time_since_epoch())
                     .count();
        }

        ~Timing()
        {
            auto end = std::chrono::duration_cast<std::chrono::microseconds>(
                     std::chrono::high_resolution_clock::now().time_since_epoch())
                     .count();
            std::cout<<name_<<" average: "<< ((end - start_) / times_) / 1000 << " ms"<<std::endl;
            
        }

    private:
        uint64_t start_;
        std::string name_;
        uint32_t times_;
};

std::unique_ptr<hipDoubleComplex []> create_random_matrix(unsigned nqubits)
{
    uint64_t size = (1ULL << nqubits);
    size = size * size;
    std::unique_ptr<hipDoubleComplex []> matrix(new hipDoubleComplex[size]);
    for (uint64_t i = 0; i < size; i++)
    {
        matrix[i].x = drand48();
        matrix[i].y = 0;
    }
    return matrix;
}

void run_random_gates(
    std::string title,
    unsigned nqubits, 
    int nIndexBits,
    const int * random_targets,
    const int * random_controls,
    int random_nTargets,
    int random_nControls  = 0,
    int random_adjoint    = 0)
{
    std::unique_ptr<hipDoubleComplex[]> random3_matrix = create_random_matrix(nqubits);
    std::stringstream ss;
    ss<<"Random"<<title<<nqubits;
    Timing timer(ss.str(), 5);
    for(int i = 0; i < 5; i++)
    {
        run_gate(random3_matrix.get(), random_nControls, random_nTargets, random_adjoint, random_targets, random_controls, nIndexBits);
    }
}

void run_null_gates(
    std::string title,
    unsigned nqubits, 
    int nIndexBits,
    const int * random_targets,
    const int * random_controls,
    int random_nTargets,
    int random_nControls  = 0,
    int random_adjoint    = 0)
{
    std::stringstream ss;
    ss<<"Random"<<title<<nqubits;
    Timing timer(ss.str(), 5);
    for(int i = 0; i < 5; i++)
    {
        run_gate(nullptr, random_nControls, random_nTargets, random_adjoint, random_targets, random_controls, nIndexBits);
    }
}


int main(int argc, char* argv[]) {

    int nIndexBits = 20;
    if (argc >= 2)
    {
        nIndexBits = atoi(argv[1]);
    }
    const double sqrt2_reverse = 1.0/sqrt(2.);
    

    const int h_targets[]  = {0};
    const int h_controls[] = {};
    const int h_nTargets   = 1;
    const int h_nControls  = 0;
    const int h_adjoint    = 0;
    hipDoubleComplex h_matrix[] = {{sqrt2_reverse, 0.0}, {sqrt2_reverse, 0.0},
                                {sqrt2_reverse, 0.0}, {-sqrt2_reverse, 0.0}};
    {
        Timing timer("HGate", 5);
        for (int i = 0; i < 5; i++)
        {
        run_gate(h_matrix, h_nControls, h_nTargets, h_adjoint, h_targets, h_controls, nIndexBits);
        }
    }

    const int cnot_targets[]  = {0};
    const int cnot_controls[] = {1};
    const int cnot_nTargets   = 1;
    const int cnot_nControls  = 1;
    const int cnot_adjoint    = 0;
    hipDoubleComplex not_matrix[] = {{0., 0.0}, {1., 0.0},
                                {1., 0.0}, {0., 0.0}};
    {
        Timing timer("CNOTGate");
        run_gate(not_matrix, cnot_nControls, cnot_nTargets, cnot_adjoint, cnot_targets, cnot_controls, nIndexBits);
    }

    const int ccnot_targets[]  = {0};
    const int ccnot_controls[] = {1, 2};
    const int ccnot_nTargets   = 1;
    const int ccnot_nControls  = 2;
    const int ccnot_adjoint    = 0;

    {
        Timing timer("CCNOTGate");
        run_gate(not_matrix, ccnot_nControls, ccnot_nTargets, ccnot_adjoint, ccnot_targets, ccnot_controls, nIndexBits);
    }

    int random_controls [] ={};
    for (unsigned nqubits = 1; nqubits <= 7; nqubits++)
    {
        int random_nTargets   = (int)nqubits;
        std::unique_ptr<int[]> random_targets (new int[random_nTargets]);
        for(int i = 0; i < (int)nqubits; i++)
        {
            random_targets[i] = nIndexBits -1 - i;
        }

        run_random_gates("High", nqubits, nIndexBits, random_targets.get(), random_controls, random_nTargets);
        run_null_gates("HighNull", nqubits, nIndexBits, random_targets.get(), random_controls, random_nTargets);
    }

    for (unsigned nqubits = 1; nqubits <= 7; nqubits++)
    {
        int random_nTargets   = (int)nqubits;
        std::unique_ptr<int[]> random_targets (new int[random_nTargets]);
        for(int i = 0; i < (int)nqubits; i++)
        {
            random_targets[i] = i;
        }

        run_random_gates("Low", nqubits, nIndexBits, random_targets.get(), random_controls, random_nTargets);
        run_null_gates("LowNull", nqubits, nIndexBits, random_targets.get(), random_controls, random_nTargets);
    }
}
